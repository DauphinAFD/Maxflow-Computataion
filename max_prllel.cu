#include<hip/hip_runtime.h>
#include<fstream>
#include<sstream>
#include<iostream>
#include<string>
#define milliseconds 1e3

# define CHECK( call )\
{\
    const hipError_t error = call;\
    if( error != hipSuccess) \
    {\
        cout << " Error " << __FILE__ << " : " << __LINE__ << endl;\
        cout << " Code : " << error << ", reason : " << hipGetErrorString(error);\
        exit(1);\
    }\
}

using namespace std;

typedef struct Vertex_info{
    int parent;
    int flow;
}Vertex_info;

void readfile(string filename,int N, int* residual_capacity){

    int source, destination, capacity;
    string line;
    stringstream ss;
    
    fstream file(filename);

    if(file.is_open()){

        while(getline(file,line)){
            
            ss.clear();
            ss.str("");
            ss.str(line);

            ss >> source >> destination >> capacity;
            residual_capacity[source*N + destination] = capacity;

        }
        file.close();
    }
}

bool sink_found (bool* frontier, int N, int sink){
    
    for(int i = N-1; i >=0 ; i--){
        if(frontier[i]){
            return i == sink ;
        }
    }
    return true;        // frontier is empty, cannot explore further
    
}

void host_set_parameters (bool *frontier, bool *change_capacity, int source, int N){

    for(int i = 0; i < N; i++ ){
        change_capacity[i] = false;
        frontier[i] = (i == source);
    }
    
}

__global__ void device_set_parameters (bool *frontier, bool *visited, bool *change_capacity, Vertex_info *Vertex, int *locks, int N, int source){

    int id = blockDim.x * blockIdx.x + threadIdx.x;

    if(id < N){
        locks[id] = 0;
        visited[id] = false;
        change_capacity[id] = false;
        frontier[id] = (id == source);
        Vertex[id].flow = INT_MAX;
    }
}

__global__ void search_path(int* residual_capacity, Vertex_info* Vertex, bool* frontier, bool* visited, int* locks, int N, int sink ){

	int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    if( id < N && frontier[id] && !frontier[sink] ){

        frontier[id] = false;
        visited[id] = true;
        int capacity; 

        Vertex_info This_Vertex = Vertex[id];
        Vertex_info* neighbour;

        for(int i = 0; i < N; i++){
            capacity = residual_capacity[id * N + i];
            if( frontier[i] || visited[i] || (capacity <= 0) ){
                continue;
            }

            if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){      // Ensure frontier is visited only once
				continue;
			}

            frontier[i] = true;

            locks[i] = 0;                           // unlock

            neighbour = Vertex + i;
            neighbour->parent = id;
            neighbour->flow = min(This_Vertex.flow, capacity);

        }
    }
}

__global__ void set_residual_capacity(Vertex_info *Vertex, bool *change_capacity, int* residual_capacity, int N, int bottleneck){

    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < N && change_capacity[id]){
        residual_capacity[Vertex[id].parent * N + id] -= bottleneck;
        residual_capacity[id * N + Vertex[id].parent] += bottleneck; 
    }    
}

int main(){

    int N = 2000;
    int *residual_capacity;
    size_t graph_size = N*N*sizeof(int);
	
    residual_capacity = (int *)malloc(graph_size);
    memset(residual_capacity, 0, graph_size); 
    
    readfile("./text/maxflow.txt",N,residual_capacity);
    
    int maxflow = 0, bottleneck = 0;
    int source = 0, sink = N-1;                             // Setting first node as Source and the last node as Sink 
    int V;
    
    bool found_path;
	bool *frontier, *change_capacity;
    int *d_residual_capacity, *d_locks;
	bool *d_frontier, *d_visited, *d_change_capacity;

    Vertex_info *Vertex, *d_Vertex;

    size_t lock_size = N*sizeof(int);
    size_t frontier_size = N*sizeof(bool);
    size_t Vertex_size = N*sizeof(Vertex_info);

    frontier = (bool *)malloc(frontier_size);
    Vertex = (Vertex_info *)malloc(Vertex_size);
    change_capacity = (bool *)malloc(frontier_size);

    CHECK( hipMalloc((void **)&d_change_capacity, frontier_size) );
    CHECK( hipMalloc((void **)&d_residual_capacity, graph_size) );
    CHECK( hipMalloc((void **)&d_frontier, frontier_size) );
    CHECK( hipMalloc((void **)&d_visited, frontier_size) );
    CHECK( hipMalloc((void **)&d_Vertex, Vertex_size) );
    CHECK( hipMalloc((void **)&d_locks, lock_size) );

    
	int threads = 512;
	int blocks = ceil(N * 1.0 /threads);

    hipMemcpy(d_residual_capacity, residual_capacity, graph_size, hipMemcpyHostToDevice);

    clock_t start_time = clock(); 

    do{                                    
        device_set_parameters<<<blocks, threads >>>(d_frontier, d_visited,d_change_capacity, d_Vertex, d_locks, N, source);
		host_set_parameters(frontier, change_capacity, source, N);
        CHECK( hipDeviceSynchronize() );

        while(!sink_found(frontier,N,sink)){

            search_path<<< blocks, threads >>>(d_residual_capacity, d_Vertex, d_frontier, d_visited, d_locks, N, sink);
            CHECK( hipDeviceSynchronize() );
            hipMemcpy(frontier, d_frontier, frontier_size, hipMemcpyDeviceToHost);
        }

        found_path = frontier[sink];

        if(!found_path){
            break;
        }
        
        hipMemcpy(Vertex, d_Vertex, Vertex_size, hipMemcpyDeviceToHost);
        
        bottleneck = Vertex[sink].flow;
        maxflow += bottleneck;

        V = sink;
        while(V!= source){
            change_capacity[V] = true;
            V = Vertex[V].parent;
        }

        hipMemcpy(d_change_capacity, change_capacity, frontier_size, hipMemcpyHostToDevice);

        set_residual_capacity<<< blocks, threads >>>(d_Vertex, d_change_capacity, d_residual_capacity, N, bottleneck);
        CHECK( hipDeviceSynchronize() );

    }while(found_path);

    double time_taken = ((double)clock() - start_time)/CLOCKS_PER_SEC * milliseconds; // in milliseconds 
    cout << endl << "\t\t ---- Ford-Fulkerson Algorithm Parallel ---- " << endl;
    cout << endl << "\t Max-Flow is : " << maxflow << endl;
	cout << " \tduration --" << time_taken << " ms "  << endl;

	CHECK( hipFree(d_residual_capacity) );
    CHECK( hipFree(d_change_capacity) );
	CHECK( hipFree(d_frontier) );
	CHECK( hipFree(d_visited) );
	CHECK( hipFree(d_Vertex) );
    CHECK( hipFree(d_locks) );
    
    free(residual_capacity);
    free(change_capacity);
	free(frontier);
	free(Vertex);

    return 0;
}